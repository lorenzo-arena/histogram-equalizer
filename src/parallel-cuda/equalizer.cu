#include "hip/hip_runtime.h"
#include "equalizer.cuh"

#include "error_checker.cuh"
#include "hsl.cuh"

extern "C" {
    #include <stdio.h>
    #include "cexception/lib/CException.h"
    #include "log.h"
    #include "errors.h"
}

#define N_BINS 500

#define BLOCK_SIZE 512

__global__ void compute_histogram(const float *image,
                                  unsigned int *bins,
                                  unsigned int num_elements)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ unsigned int bins_s[];
    for (unsigned int binIdx = threadIdx.x; binIdx < N_BINS; binIdx += blockDim.x)
    {
        bins_s[binIdx] = 0;
    }

    __syncthreads();

    for (unsigned int i = tid; i < num_elements; i += blockDim.x * gridDim.x)
    {
        atomicAdd(&(bins_s[(unsigned int)__float2int_rn(image[i] * (N_BINS - 1))]), 1);
    }

    __syncthreads();

    for (unsigned int binIdx = threadIdx.x; binIdx < N_BINS; binIdx += blockDim.x)
    {
        atomicAdd(&(bins[binIdx]), bins_s[binIdx]);
    }
}

__global__ void convert_rgb_to_hsl(const uint8_t *rgb_image,
                                   hsl_image_t hsl_image,
                                   unsigned int num_elements)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < num_elements)
    {
        const uint8_t *pixel_offset = &rgb_image[tid * 3];

        rgb_pixel_t rgb_pixel = {
            .r = pixel_offset[0],
            .g = pixel_offset[1],
            .b = pixel_offset[2]
        };

        hsl_pixel_t hsl_pixel = { .h = 0, .s = 0, .l = 0 };

        rgb_to_hsl(rgb_pixel, &hsl_pixel);

        hsl_image.h[tid] = hsl_pixel.h;
        hsl_image.s[tid] = hsl_pixel.s;
        hsl_image.l[tid] = hsl_pixel.l;
    }
}

__global__ void convert_hsl_to_rgb(const hsl_image_t hsl_image,
                                   uint8_t *rgb_image,
                                   unsigned int num_elements)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < num_elements)
    {
        uint8_t *pixel_offset = rgb_image + (tid * 3);

        rgb_pixel_t rgb_pixel = { .r = 0, .g = 0, .b = 0 };

        hsl_pixel_t hsl_pixel = {
            .h = hsl_image.h[tid],
            .s = hsl_image.s[tid],
            .l = hsl_image.l[tid]
        };

        hsl_to_rgb(hsl_pixel, &rgb_pixel);

        pixel_offset[0] = rgb_pixel.r;
        pixel_offset[1] = rgb_pixel.g;
        pixel_offset[2] = rgb_pixel.b;
    }
}

__global__ void compute_cdf(unsigned int *input, unsigned int *output, int input_size)
{
    __shared__ unsigned int sh_out[BLOCK_SIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < input_size)
    {
        sh_out[threadIdx.x] = input[tid];
    }

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        __syncthreads();
        if(threadIdx.x >= stride)
        {
            sh_out[threadIdx.x] += sh_out[threadIdx.x - stride];
        }
    }

    __syncthreads();

    if (tid < input_size)
    {
        output[tid] = sh_out[threadIdx.x];
    }
}

int equalize(uint8_t *input, unsigned int width, unsigned int height, uint8_t **output)
{
    CEXCEPTION_T e = NO_ERROR;

    int blocksPerGrid = 0;

    uint8_t *d_rgb_image = NULL;
    uint8_t *d_output_image = NULL;
    unsigned int *d_histogram = NULL;
    unsigned int *d_cdf = NULL;

    hsl_image_t d_hsl_image = {
        .h = NULL,
        .s = NULL,
        .l = NULL
    };

    Try {
        // Allocate memory for the image on the device
        gpuErrorCheck( hipMalloc((void**)&d_rgb_image, 3 * width * height * sizeof(uint8_t)) );
        gpuErrorCheck( hipMemcpy(d_rgb_image, input, 3 * width * height, hipMemcpyHostToDevice) );

        gpuErrorCheck( hipMalloc((void**)&(d_hsl_image.h), width * height * sizeof(int)) );
        gpuErrorCheck( hipMalloc((void**)&(d_hsl_image.s), width * height * sizeof(float)) );
        gpuErrorCheck( hipMalloc((void**)&(d_hsl_image.l), width * height * sizeof(float)) );

        // Allocate memory for the output
        *output = (uint8_t *)calloc(3 * width * height, sizeof(uint8_t));

        if(NULL == (*output))
        {
            Throw(UNALLOCATED_MEMORY);
        }

        gpuErrorCheck( hipMalloc((void**)&d_output_image, 3 * width * height * sizeof(uint8_t)) );

        gpuErrorCheck( hipMalloc((void**)&d_histogram, N_BINS * sizeof(unsigned int)) );
        gpuErrorCheck( hipMalloc((void**)&d_cdf, N_BINS * sizeof(unsigned int)) );

        // **************************************
        // STEP 1 - convert every pixel from RGB to HSL
        blocksPerGrid = ((width * height) + BLOCK_SIZE - 1) / BLOCK_SIZE;
        convert_rgb_to_hsl<<<blocksPerGrid, BLOCK_SIZE>>>(d_rgb_image, d_hsl_image, width * height);

        // **************************************
        // STEP 2 - compute the histogram of the luminance for each pixel
        blocksPerGrid = 30;
        compute_histogram<<<blocksPerGrid, BLOCK_SIZE, N_BINS * sizeof(unsigned int)>>>(d_hsl_image.l, d_histogram, (width * height));

        // **************************************
        // STEP 3 - compute the cumulative distribution function by applying the parallelized
        // version of the scan algorithm
        blocksPerGrid = (N_BINS + BLOCK_SIZE - 1) / BLOCK_SIZE;
        compute_cdf<<<blocksPerGrid, BLOCK_SIZE>>>(d_histogram, d_cdf, N_BINS);

        // **************************************
        // STEP 4 - compute the normalized cumulative distribution function

        // **************************************
        // STEP 5 - apply the normalized CDF to the luminance for each pixel

        // **************************************
        // STEP 6 - convert each HSL pixel back to RGB
        blocksPerGrid = ((width * height) + BLOCK_SIZE - 1) / BLOCK_SIZE;
        convert_hsl_to_rgb<<<blocksPerGrid, BLOCK_SIZE>>>(d_hsl_image, d_output_image, width * height);

        // Copy the result back from the device
        gpuErrorCheck( hipMemcpy(*output, d_output_image, 3 * width * height, hipMemcpyDeviceToHost) );
    } Catch(e) {
        log_error("Caught exception %d while equalizing image!", e);
    }

    hipFree(d_rgb_image);
    hipFree(d_output_image);
    hipFree(d_histogram);
    hipFree(d_cdf);
    hipFree(d_hsl_image.h);
    hipFree(d_hsl_image.s);
    hipFree(d_hsl_image.l);

    return e;
}